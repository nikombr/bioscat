#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
extern "C" {
#include "../../lib/Segment.h"
#include "../../lib/RealMatrix.h"
using namespace std;

double H02_real(double x) {
    // Computes real part of Hankel function of order zero and second kind
    int n = 0;
    double Jn = jn(n, x); // Compute Bessel functions of the first (Jn) 
    return Jn;
}

double H02_imag(double x) {
    // Computes imaginary part of Hankel function of order zero and second kind
    int n = 0;
    double Yn = yn(n, x); // Compute Bessel functions of the second (Yn) kind
    return -Yn;
}

double H12_real(double x) {
    // Computes real part of Hankel function of order one and second kind
    int n = 1;
    double Jn = jn(n, x); // Compute Bessel functions of the first (Jn) 
    return Jn; 
}

double H12_imag(double x) {
    // Computes imaginary part of Hankel function of order one and second kind
    int n = 1;
    double Yn = yn(n, x); // Compute Bessel functions of the second (Yn) kind
    return -Yn;
}

void Segment::computeScatteredFieldMatrices(RealMatrix x, RealMatrix y, bool far_field_approximation) {
    
    int rows = y.rows;
    int cols = y_int.rows;
    double abs_int, abs_int_ref, xdiff, ydiff, ydiff_ref, H_real, H_imag, H_real_ref, H_imag_ref, val;

    // Determine which matrices to be allocated
    bool Ex_bool = scenario == 1 ? false : true;
    bool Ey_bool = scenario == 1 ? false : true;
    bool Ez_bool = scenario == 1 ? true  : false;
    bool Hx_bool = scenario == 1 ? true  : false;
    bool Hy_bool = scenario == 1 ? true  : false;
    bool Hz_bool = scenario == 1 ? false : true;

    E_scat_matrix = Field(rows, cols, Ex_bool, Ey_bool, Ez_bool);
    H_scat_matrix = Field(rows, cols, Hx_bool, Hy_bool, Hz_bool);

    if (scenario == 1) {
        for (int r = 0; r < rows; r++) {
            for (int c = 0; c < cols; c++) {

                // Get data
                xdiff       = x.getHostValue(r) - x_int.getHostValue(c);
                ydiff       = y.getHostValue(r) - y_int.getHostValue(c);
                ydiff_ref   = y.getHostValue(r) + y_int.getHostValue(c);
                abs_int     = std::sqrt(xdiff*xdiff + ydiff*ydiff);
                abs_int_ref = std::sqrt(xdiff*xdiff + ydiff_ref*ydiff_ref);

                // Compute first Hankel functions
                H_real     = H02_real(constants.k0*abs_int);
                H_real_ref = H02_real(constants.k0*abs_int_ref);
                H_imag     = H02_imag(constants.k0*abs_int);
                H_imag_ref = H02_imag(constants.k0*abs_int_ref);
                
                val = H_real + constants.Gamma_ref * H_real_ref;
                E_scat_matrix.z.setHostRealValue(r, c, val);
                val = H_imag + constants.Gamma_ref * H_imag_ref;
                E_scat_matrix.z.setHostImagValue(r, c, val);

                // Compute second Hankel functions
                H_real     = H12_real(constants.k0*abs_int);
                H_real_ref = H12_real(constants.k0*abs_int_ref);
                H_imag     = H12_imag(constants.k0*abs_int);
                H_imag_ref = H12_imag(constants.k0*abs_int_ref);

                val = 1/constants.eta0 * (1/abs_int      * H_imag     * ydiff + \
                     constants.Gamma_ref * 1/abs_int_ref * H_imag_ref * ydiff_ref);
                H_scat_matrix.x.setHostRealValue(r, c, val);
                val = -1/constants.eta0 * (1/abs_int     * H_real     * ydiff + \
                     constants.Gamma_ref * 1/abs_int_ref * H_real_ref * ydiff_ref);
                H_scat_matrix.x.setHostImagValue(r, c, val);

                val = -1/constants.eta0 * xdiff * (1/abs_int      * H_imag      + \
                             constants.Gamma_ref * 1/abs_int_ref  * H_imag_ref);
                H_scat_matrix.y.setHostRealValue(r, c, val);
                val = 1/constants.eta0 * xdiff * (1/abs_int     * H_real      + \
                            constants.Gamma_ref * 1/abs_int_ref * H_real_ref);
                H_scat_matrix.y.setHostImagValue(r, c, val);
            }
        }
        /*printf("\nEz:\n");
        for (int r = 0; r < rows; r++) {
            printf("%e + i(%e)\n",E_scat_matrix.z.getHostRealValue(r,0),E_scat_matrix.z.getHostImagValue(r,0));

        }
        printf("\nHx:\n");
        for (int r = 0; r < rows; r++) {
            printf("%e + i(%e)\n",H_scat_matrix.x.getHostRealValue(r,0),H_scat_matrix.x.getHostImagValue(r,0));

        }
        printf("\nHy:\n");
        for (int r = 0; r < rows; r++) {
            printf("%e + i(%e)\n",H_scat_matrix.y.getHostRealValue(r,0),H_scat_matrix.y.getHostImagValue(r,0));

        }*/

    }
    else if (scenario == 2) {

    }
    else {
        printf("Please input 1 or 2 for the scenario!\n");
    }

}

void Segment::computeInteriorFieldMatrices(RealMatrix x, RealMatrix y) {
    
    int rows = y.rows;
    int cols = y_ext.rows;
    double abs_ext, xdiff, ydiff, H_real, H_imag, val;

    // Determine which matrices to be allocated
    bool Ex_bool = scenario == 1 ? false : true;
    bool Ey_bool = scenario == 1 ? false : true;
    bool Ez_bool = scenario == 1 ? true  : false;
    bool Hx_bool = scenario == 1 ? true  : false;
    bool Hy_bool = scenario == 1 ? true  : false;
    bool Hz_bool = scenario == 1 ? false : true;

    E_int_matrix = Field(rows, cols, Ex_bool, Ey_bool, Ez_bool);
    H_int_matrix = Field(rows, cols, Hx_bool, Hy_bool, Hz_bool);

    if (scenario == 1) {
        for (int r = 0; r < rows; r++) {
            for (int c = 0; c < cols; c++) {

                // Get data
                xdiff   = x.getHostValue(r) - x_ext.getHostValue(c);
                ydiff   = y.getHostValue(r) - y_ext.getHostValue(c);
                abs_ext = std::sqrt(xdiff*xdiff + ydiff*ydiff);

                // Compute first Hankel functions
                H_real = H02_real(constants.k1*abs_ext);
                H_imag = H02_imag(constants.k1*abs_ext);
                
                val = H_real;
                E_int_matrix.z.setHostRealValue(r, c, val);
                val = H_imag;
                E_int_matrix.z.setHostImagValue(r, c, val);

                // Compute second Hankel functions
                H_real = H12_real(constants.k1*abs_ext);
                H_imag = H12_imag(constants.k1*abs_ext);

                val =   constants.n1/constants.eta0 * 1/abs_ext * ydiff * H_imag;
                H_int_matrix.x.setHostRealValue(r, c, val);
                val = - constants.n1/constants.eta0 * 1/abs_ext * ydiff * H_real;
                H_int_matrix.x.setHostImagValue(r, c, val);

                val = -constants.n1/constants.eta0 * 1/abs_ext * xdiff * H_imag;
                H_int_matrix.y.setHostRealValue(r, c, val);
                val =  constants.n1/constants.eta0 * 1/abs_ext * xdiff * H_real;
                H_int_matrix.y.setHostImagValue(r, c, val);
            }
        }
        /*printf("\nEz:\n");
        for (int r = 0; r < rows; r++) {
            printf("%e + i(%e)\n",E_int_matrix.z.getHostRealValue(r,0),E_int_matrix.z.getHostImagValue(r,0));

        }
        printf("\nHx:\n");
        for (int r = 0; r < rows; r++) {
            printf("%e + i(%e)\n",H_int_matrix.x.getHostRealValue(r,0),H_int_matrix.x.getHostImagValue(r,0));

        }
        printf("\nHy:\n");
        for (int r = 0; r < rows; r++) {
            printf("%e + i(%e)\n",H_int_matrix.y.getHostRealValue(r,0),H_int_matrix.y.getHostImagValue(r,0));

        }*/

    }
    else if (scenario == 2) {

    }
    else {
        printf("Please input 1 or 2 for the scenario!\n");
    }

}


}