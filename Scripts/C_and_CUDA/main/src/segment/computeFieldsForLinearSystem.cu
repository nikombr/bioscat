#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
extern "C" {
#include "../../lib/Segment.h"
#include "../../lib/RealMatrix.h"
using namespace std;

void Segment::computeFieldsForLinearSystem() {

    
    computeIncidentFieldVectors(y_test);
    printf("hej\n");
    computeReflectedFieldVectors(y_test);
    computeScatteredFieldMatrices(x_test, y_test, false);
    computeInteriorFieldMatrices(x_test, y_test);

    //x_test.free();
    //y_test.free();

}

}