#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
extern "C" {
#include "../lib/ComplexMatrix.h"
using namespace std;

ComplexMatrix::ComplexMatrix(int rows, int cols) {

    // Save input parameters
    this->rows       = rows;
    this->cols       = cols;

    // Allocate vectors on host
    hipHostMalloc((void **) &real_h,    rows*cols*sizeof(double));
    hipHostMalloc((void **) &imag_h, rows*cols*sizeof(double));

    // Allocate vectors on device
    hipMalloc((void **) &real_d,    rows*cols*sizeof(double));
    hipMalloc((void **) &imag_d, rows*cols*sizeof(double));
}

ComplexMatrix::ComplexMatrix(int rows) {

    // Save input parameters
    this->rows       = rows;
    this->cols       = 1;

    // Allocate vectors on host
    hipHostMalloc((void **) &real_h,    rows*cols*sizeof(double));
    hipHostMalloc((void **) &imag_h, rows*cols*sizeof(double));

    // Allocate vectors on device
    hipMalloc((void **) &real_d,    rows*cols*sizeof(double));
    hipMalloc((void **) &imag_d, rows*cols*sizeof(double));
}

void ComplexMatrix::free() {
    
    // Free on host
    hipHostFree(real_h);
    hipHostFree(imag_h);

    // Free on device
    hipFree(real_d);
    hipFree(imag_d);

}

void ComplexMatrix::toHost() {

    // Send from device to host
    hipMemcpy(real_h,    real_d,    rows * cols * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(imag_h, imag_d, rows * cols * sizeof(double), hipMemcpyDeviceToHost);

}

void ComplexMatrix::toDevice() {

    // Send from host to device
    hipMemcpy(real_d,    real_h,    rows * cols * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(imag_d, imag_h, rows * cols * sizeof(double), hipMemcpyHostToDevice);
    
}

void ComplexMatrix::setHostRealValue(int r, double val) {
    real_h[r] = val;
}

void ComplexMatrix::setHostRealValue(int r, int c, double val) {
    real_h[r*cols + c] = val;
}

__device__ void ComplexMatrix::setDeviceRealValue(int r, double val) {
    real_d[r] = val;
}

__device__ void ComplexMatrix::setDeviceRealValue(int r, int c, double val) {
    real_d[r*cols + c] = val;
}

double ComplexMatrix::getHostRealValue(int r) {
    return real_h[r];
}

double ComplexMatrix::getHostRealValue(int r, int c) {
    return real_h[r*cols + c];
}

__device__ double ComplexMatrix::getDeviceRealValue(int r) {
    return real_d[r];
}

__device__ double ComplexMatrix::getDeviceRealValue(int r, int c) {
    return real_d[r*cols + c];
}

void ComplexMatrix::setHostImagValue(int r, double val) {
    imag_h[r] = val;
}

void ComplexMatrix::setHostImagValue(int r, int c, double val) {
    imag_h[r*cols + c] = val;
}

__device__ void ComplexMatrix::setDeviceImagValue(int r, double val) {
    imag_d[r] = val;
}

__device__ void ComplexMatrix::setDeviceImagValue(int r, int c, double val) {
    imag_d[r*cols + c] = val;
}

double ComplexMatrix::getHostImagValue(int r) {
    return imag_h[r];
}

double ComplexMatrix::getHostImagValue(int r, int c) {
    return imag_h[r*cols + c];
}

__device__ double ComplexMatrix::getDeviceImagValue(int r) {
    return imag_d[r];
}

__device__ double ComplexMatrix::getDeviceImagValue(int r, int c) {
    return imag_d[r*cols + c];
}

}