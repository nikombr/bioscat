#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
extern "C" {
#include "../lib/ComplexMatrix.h"
using namespace std;

ComplexMatrix::ComplexMatrix(int rows, int cols) {

    // Save input parameters
    this->rows       = rows;
    this->cols       = cols;

    // Allocate vectors on host
    hipHostMalloc((void **) &real_h,    rows*cols*sizeof(double));
    hipHostMalloc((void **) &imag_h, rows*cols*sizeof(double));

    // Allocate vectors on device
    hipMalloc((void **) &real_d,    rows*cols*sizeof(double));
    hipMalloc((void **) &imag_d, rows*cols*sizeof(double));
}

ComplexMatrix::ComplexMatrix(int rows) {

    // Save input parameters
    this->rows       = rows;
    this->cols       = 1;

    // Allocate vectors on host
    hipHostMalloc((void **) &real_h,    rows*cols*sizeof(double));
    hipHostMalloc((void **) &imag_h, rows*cols*sizeof(double));

    // Allocate vectors on device
    hipMalloc((void **) &real_d,    rows*cols*sizeof(double));
    hipMalloc((void **) &imag_d, rows*cols*sizeof(double));
}

void ComplexMatrix::free() {
    
    // Free on host
    hipError_t err = hipHostFree(real_h);
    if (err != hipSuccess) {
    std::cerr << "Failed to free memory: " << hipGetErrorString(err) << std::endl;
}
     err = hipHostFree(imag_h);

    if (err != hipSuccess) {
    std::cerr << "Failed to free memory: " << hipGetErrorString(err) << std::endl;
}

    // Free on device
    hipFree(real_d);
    hipFree(imag_d);

}

void ComplexMatrix::toHost() {

    // Send from device to host
    hipMemcpy(real_h,    real_d,    rows * cols * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(imag_h, imag_d, rows * cols * sizeof(double), hipMemcpyDeviceToHost);

}

void ComplexMatrix::toDevice() {

    // Send from host to device
    hipMemcpy(real_d,    real_h,    rows * cols * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(imag_d, imag_h, rows * cols * sizeof(double), hipMemcpyHostToDevice);
    
}

void ComplexMatrix::setHostRealValue(int r, double val) {
    real_h[r] = val;
}

void ComplexMatrix::setHostRealValue(int r, int c, double val) {
    real_h[r*cols + c] = val;
}

double ComplexMatrix::getHostRealValue(int r) {
    return real_h[r];
}

double ComplexMatrix::getHostRealValue(int r, int c) {
    return real_h[r*cols + c];
}


void ComplexMatrix::setHostImagValue(int r, double val) {
    imag_h[r] = val;
}

void ComplexMatrix::setHostImagValue(int r, int c, double val) {
    imag_h[r*cols + c] = val;
}


double ComplexMatrix::getHostImagValue(int r) {
    return imag_h[r];
}

double ComplexMatrix::getHostImagValue(int r, int c) {
    return imag_h[r*cols + c];
}


void ComplexMatrix::dumpResult(const char * filename) {
    if (cols == 1) {
        FILE *file;
        file = fopen(filename, "w");
        if (file == NULL) {
            perror("Error opening file");
            return;
        }
        for (int r = 0; r < rows; r++) {
            fprintf(file, "%e\t%e\n", getHostRealValue(r), getHostImagValue(r));
        }
        fclose(file);
    }
    else printf("We do not support saving complex matrices with several columns.\n");
}


}