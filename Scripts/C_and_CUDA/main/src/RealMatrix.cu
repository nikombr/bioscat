
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
extern "C" {
#include "../lib/RealMatrix.h"
using namespace std;

RealMatrix::RealMatrix(int rows) {
     // Save input parameters
    this->rows       = rows;
    this->cols       = 1;

    // Allocate vectors on host
    hipHostMalloc((void **) &val_h,    rows*cols*sizeof(double));

    // Allocate vectors on device
    hipMalloc((void **) &val_d,    rows*cols*sizeof(double));

    val_h[0] = 1;
}

RealMatrix::RealMatrix(int rows, int cols) {

    // Save input parameters
    this->rows       = rows;
    this->cols       = cols;

    // Allocate vectors on host
    hipHostMalloc((void **) &val_h,    rows*cols*sizeof(double));

    // Allocate vectors on device
    hipMalloc((void **) &val_d,    rows*cols*sizeof(double));

    val_h[0] = 1;
}

void RealMatrix::free() {
    
    // Free on host
    hipHostFree(val_h);

    // Free on device
    hipFree(val_d);

    printf("Destructed real matrices!\n");

}

void RealMatrix::toHost() {

    // Send from device to host
    hipMemcpy(val_h,    val_d,    rows * cols * sizeof(double), hipMemcpyDeviceToHost);

}

void RealMatrix::toDevice() {

    // Send from host to device
    hipMemcpy(val_d,    val_h,    rows * cols * sizeof(double), hipMemcpyHostToDevice);
    
}

void RealMatrix::setHostValue(int r, double val) {
    val_h[r] = val;
}

void RealMatrix::setHostValue(int r, int c, double val) {
    val_h[r*cols + c] = val;
}

__device__ void RealMatrix::setDeviceValue(int r, double val) {
    val_h[r] = val;
}

__device__ void RealMatrix::setDeviceValue(int r, int c, double val) {
    val_h[r*cols + c] = val;
}

double RealMatrix::getHostValue(int r) {
    return val_h[r];
}

double RealMatrix::getHostValue(int r, int c) {
    return val_h[r*cols + c];
}

__device__ double RealMatrix::getDeviceValue(int r) {
    return val_h[r];
}

__device__ double RealMatrix::getDeviceValue(int r, int c) {
    return val_h[r*cols + c];
}


}