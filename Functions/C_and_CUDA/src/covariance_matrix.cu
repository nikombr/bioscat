#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
extern "C" {
#include "../lib/GaussianProcess.h"

__device__ __host__ double squared_exponential(double* a, double* b, int dim,  double* hyper) {
    double tau = hyper[0];
    double ell = hyper[1];
    double temp = 0;
    for (int i = 0; i < dim; i++) {
        temp += (a[i]-b[i]) * (a[i]-b[i]);
    }

    return tau*tau*exp(-temp/(2*ell*ell));

}


#define kfunc squared_exponential

__global__ void covariance_matrix_device_1d(double ** Sigma, double* x, int dim, double* hyper, int n) {
    double a[1], b[1];
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    if (k < n && i >= k && i < n) {
        a[0] = x[k];
        b[0] = x[i];
        Sigma[i][k] = kfunc(a, b, dim, hyper);
    }
}

__global__ void covariance_matrix_device_2d(double ** Sigma, double* x, double* y, int dim, double* hyper, int n) {
    
    double a[2], b[2];
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    if (k < n && i >= k && i < n) {
        a[0] = x[k];
        b[0] = x[i];
        a[1] = y[k];
        b[1] = y[i];
        Sigma[i][k] = kfunc(a, b, dim, hyper);
    }
}



void GaussianProcess::covariance_matrix() {

    if (device) {
        if (dim == 1) {
            // Blocks and threads
            dim3 dimBlock(32,32);
            dim3 dimGrid((n+dimBlock.x-1)/dimBlock.x,(n+dimBlock.y-1)/dimBlock.y);
            // Call kernel
            covariance_matrix_device_1d<<<dimGrid, dimBlock>>>(M_d, x_d, dim, hyper_d, n);
            hipDeviceSynchronize();
        }
        else if (dim == 2) {
            // Blocks and threads
            dim3 dimBlock(32,32);
            dim3 dimGrid((n+dimBlock.x-1)/dimBlock.x,(n+dimBlock.y-1)/dimBlock.y);
            // Call kernel
            covariance_matrix_device_2d<<<dimGrid, dimBlock>>>(M_d, x_d, y_d, dim, hyper_d, n);
            hipDeviceSynchronize();

        }


    }
    else {

        if (dim == 1) {
            double a[1], b[1];
            for (int k = 0; k < n; k++) {
                for (int i = k; i < n; i++) {
                    a[0] = x_h[k];
                    b[0] = x_h[i];
                    
                    M_h[i][k] = kfunc(a, b, dim, hyper_h);
                }
            }
        }
        else if (dim == 2) {
            double a[2], b[2];
            for (int k = 0; k < n; k++) {
                for (int i = k; i < n; i++) {
                    
                    a[0] = x_h[k];
                    b[0] = x_h[i];
                    a[1] = y_h[k];
                    b[1] = y_h[i];

                    M_h[i][k] = kfunc(a, b, dim, hyper_h);
                }
            }
        }


    }

}

}